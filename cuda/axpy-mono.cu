#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void daxpy_mono_(int n, double a, double *x, double *y)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < n) {
        y[tid] += a * x[tid];
    }
}

__global__ void saxpy_mono_(int n, float a, float *x, float *y)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < n) {
        y[tid] += a * x[tid];
    }
}

int main(void)
{
    int i;
    const int n = 10000;
    double a = 3.4;
    double x[n], y[n], y_ref[n];
    double *x_, *y_;

    // initialise data and calculate reference values on CPU
    for (i=0; i < n; i++) {
        x[i] = sin(i) * 2.3;
        y[i] = cos(i) * 1.1;
        y_ref[i] = a * x[i] + y[i];
    }

    // allocate + copy initial values
    hipMalloc((void **) &x_, sizeof(double) * n);
    hipMalloc((void **) &y_, sizeof(double) * n);
    hipMemcpy(x_, x, sizeof(double) * n, hipMemcpyHostToDevice);
    hipMemcpy(y_, y, sizeof(double) * n, hipMemcpyHostToDevice);

    // calculate gridsize for a one-pass kernel launch
    int blockSize = 256;
    int gridSize = ((int) (n / (blockSize * 32)) + 1) * 32;
    printf("<<<gridSize,blockSize>>> = <<<%d,%d>>>\n", gridSize, blockSize);

    // calculate axpy on GPU using a monolithic kernel
    daxpy_mono_<<<gridSize,blockSize>>>(n, a, x_, y_);

    // copy result back to host and print with reference
    printf("  initial: %f %f %f %f ... %f %f\n",
            y[0], y[1], y[2], y[3], y[n-2], y[n-1]);
    hipMemcpy(&y, y_, sizeof(double) * n, hipMemcpyDeviceToHost);
    printf("reference: %f %f %f %f ... %f %f\n",
            y_ref[0], y_ref[1], y_ref[2], y_ref[3], y_ref[n-2], y_ref[n-1]);
    printf("   result: %f %f %f %f ... %f %f\n",
            y[0], y[1], y[2], y[3], y[n-2], y[n-1]);

    return 0;
}
