#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void copy_(double *src, double *tgt, int n, int m, int offset)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;

    for (; tid < m; tid += stride) {
        if (tid + offset < n)
            tgt[tid] = src[tid + offset];
    }
}


int main(void)
{
    int i;
    const int n = 10000;
    const int m = 1000;
    int position = 22;
    double x[n], y[m], y_ref[m];
    double *x_, *y_;

    dim3 blocks(32, 1, 1);
    dim3 threads(256, 1, 1);

    // initialise data
    for (i=0; i < n; i++) {
        x[i] = (double) i / 1000.0;
    }
    for (i=0; i < m; i++) {
        y[i] = 0.0;
    }
    // copy reference values
    for (i=0; i < m; i++) {
        y_ref[i] = x[i + position];
    }

    // allocate + copy initial values
    hipMalloc((void **) &x_, sizeof(double) * n);
    hipMalloc((void **) &y_, sizeof(double) * m);
    hipMemcpy(x_, x, sizeof(double) * n, hipMemcpyHostToDevice);
    hipMemcpy(y_, y, sizeof(double) * m, hipMemcpyHostToDevice);

    // copy subarray on GPU
    copy_<<<blocks, threads>>>(x_, y_, n, m, position);

    // copy result back to host and print with reference
    printf("  initial: %f %f %f %f ... %f %f\n",
            y[0], y[1], y[2], y[3], y[m-2], y[m-1]);
    hipMemcpy(y, y_, sizeof(double) * m, hipMemcpyDeviceToHost);
    printf("reference: %f %f %f %f ... %f %f\n",
            y_ref[0], y_ref[1], y_ref[2], y_ref[3], y_ref[m-2], y_ref[m-1]);
    printf("   result: %f %f %f %f ... %f %f\n",
            y[0], y[1], y[2], y[3], y[m-2], y[m-1]);

    return 0;
}
