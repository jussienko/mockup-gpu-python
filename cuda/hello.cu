#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello_(int n, int *tag)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;

    for (; tid < n; tid += stride) {
        tag[tid] = 1;
    }
}

__global__ void hello_single_(int n, int *tag)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < n)
        tag[tid] = 1;
}

__global__ void hello2D_(int n, int2 dim, int *tag)
{
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    int stridex = gridDim.x * blockDim.x;
    int stridey = gridDim.y * blockDim.y;
    int gid;

    for (; tidy < dim.x; tidy += stridey) {
        for (; tidx < dim.y; tidx += stridex) {
            gid = stridex * tidy + tidx;
            if (gid < n)
                tag[gid] = 1;
        }
    }
}

__global__ void hello2D_single_(int n, int *tag)
{
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    int gid = gridDim.x * blockDim.x * tidy + tidx;

    if (gid < n)
        tag[gid] = 1;
}

/*
   Find continuous ranges of non-zero values in buffer. Stores start and
   end indeces in span.
*/
void range_finder(int n, int *buffer, int *span)
{
    int on = 0;
    int j = 0;
    for (int i=0; i < n; i++) {
        span[i] = -1;
        if (buffer[i]) {
            if (!on) {
                span[j] = i;
                j++;
            }
            on = 1;
        } else if (on) {
            span[j] = i - 1;
            j++;
            on = 0;
        }
    }
    if (on) {
        span[j] = n - 1;
    }
}

/*
   Print hellos from non-zero tags, coalescing continuous ranges.
*/
void print_hello(int n, int *tag)
{
    int start, end;
    int span[n];

    range_finder(n, tag, (int *) span);
    int i = 0;
    while (i < n && span[i] != -1) {
        start = span[i];
        end = span[i+1];
        i += 2;
        if (start == end)
            printf("Hello from %d\n", start);
        else
            printf("Hello from {%d..%d}\n", start, end);
    }
}


int main(void)
{
    int i;
    const int maxsize = 65536;
    const int n = 10000;
    int2 size2 = {100,100};
    int tag[maxsize];
    int *tag_;

    dim3 blocks(32);
    dim3 threads(256);
    dim3 blocks2(32,32);
    dim3 threads2(16,16);

    hipMalloc((void **) &tag_, sizeof(int) * maxsize);

    // initialise tags
    for (i=0; i < maxsize; i++) {
        tag[i] = 0;
    }
    hipMemcpy(tag_, tag, sizeof(int) * maxsize, hipMemcpyHostToDevice);

    // simple 1D kernel (single thread, single operation)
    hello_single_<<<blocks, threads>>>(n, tag_);

    // print out the hellos
    hipMemcpy(tag, tag_, sizeof(int) * maxsize, hipMemcpyDeviceToHost);
    printf("\nSINGLE\n");
    print_hello(n, (int *) tag);

    // initialise tags
    for (i=0; i < maxsize; i++) {
        tag[i] = 0;
    }
    hipMemcpy(tag_, tag, sizeof(int) * maxsize, hipMemcpyHostToDevice);

    // flexible 1D kernel (single thread, multiple operations)
    hello_<<<blocks, threads>>>(n, tag_);

    // print out the hellos
    hipMemcpy(tag, tag_, sizeof(int) * maxsize, hipMemcpyDeviceToHost);
    printf("\n1D\n");
    print_hello(n, (int *) tag);

    // initialise tags
    for (i=0; i < maxsize; i++) {
        tag[i] = 0;
    }
    hipMemcpy(tag_, tag, sizeof(int) * maxsize, hipMemcpyHostToDevice);

    // simple 2D kernel
    hello2D_single_<<<blocks, threads>>>(n, tag_);

    // print out the hellos
    hipMemcpy(tag, tag_, sizeof(int) * maxsize, hipMemcpyDeviceToHost);
    printf("\n2D SINGLE\n");
    print_hello(n, (int *) tag);

    // initialise tags
    for (i=0; i < maxsize; i++) {
        tag[i] = 0;
    }
    hipMemcpy(tag_, tag, sizeof(int) * maxsize, hipMemcpyHostToDevice);

    // flexible 2D kernel
    hello2D_<<<blocks2, threads2>>>(n, size2, tag_);

    // print out the hellos
    hipMemcpy(tag, tag_, sizeof(int) * maxsize, hipMemcpyDeviceToHost);
    printf("\n2D\n");
    print_hello(n, (int *) tag);

    return 0;
}
