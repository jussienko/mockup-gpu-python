#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void sum_(int n, double *x, double *y)
{
    int tid = threadIdx.x;
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    extern __shared__ double buffer[];
    int i;

    double sum = 0.0f;
    for (i = gid; i < n; i += stride) {
        sum += x[i];
    }
    buffer[tid] = sum;
    __syncthreads();

    i = blockDim.x / 2;
    while (i > 0) {
        if (tid < i)
            buffer[tid] += buffer[tid + i];
        i = i / 2;
        __syncthreads();
    }

    if (tid == 0)
        y[blockIdx.x] = buffer[0];
}

int main(void)
{
    int i;
    const int blocks = 32;
    const int threads = 256;
    const int n = 10000;
    double x[n];
    double z, z_ref;
    double *x_, *z_, *buffer_;
    double partial[blocks];

    // initialise data and calculate reference values on CPU
    z_ref = 0.0f;
    for (i=0; i < n; i++) {
        x[i] = sin(i) * 2.3;
        z_ref += x[i];
    }

    // allocate + copy initial values
    hipMalloc((void **) &x_, sizeof(double) * n);
    hipMalloc((void **) &z_, sizeof(double));
    hipMalloc((void **) &buffer_, sizeof(double) * blocks);
    hipMemcpy(x_, x, sizeof(double) * n, hipMemcpyHostToDevice);

    // calculate partial sums on GPU
    sum_<<<blocks,threads,threads*sizeof(double)>>>(n, x_, buffer_);
    hipMemcpy(&partial, buffer_, sizeof(double) * blocks, hipMemcpyDeviceToHost);
    double d = 0.0;
    for (i=0; i < blocks; i++) {
        d += partial[i];
    }
    printf("sum(parti): %f\n", d);
    printf("   partial: %f %f %f %f %f ...\n",
            partial[0], partial[1], partial[2], partial[3], partial[4]);
    // reduce partial sums
    sum_<<<1,blocks,blocks*sizeof(double)>>>(blocks, buffer_, z_);

    // copy result back to host and print with reference
    hipMemcpy(&z, z_, sizeof(double), hipMemcpyDeviceToHost);
    printf(" reference: %f\n    result: %f\n", z_ref, z);

    return 0;
}
